#include "hip/hip_runtime.h"
#include<iostream>
#include<stdlib.h>

#include "hip/hip_runtime.h"
#include ""
#include <assert.h>

int number_count = 640;
const int allocation_size = number_count * sizeof(int);

void* cpu_p;
void* gpu_p;


void cpuSetNumbers()
{
	int* cpu_int32 = (int*)cpu_p;

	for (int i = 0; i < number_count; i++)
		cpu_int32[i] = i;

}

int cpuAlloc()							// return 1 -> failed 
{
	cpu_p = malloc(allocation_size);

	if (cpu_p != nullptr)
		return 0;
	return 1;
}

hipError_t gpuAlloc()
{
	hipError_t result = hipMalloc(&gpu_p, allocation_size);
	return result;
}


hipError_t cpuMemoryToGpuMemory()
{
	hipError_t result = hipMemcpy(gpu_p, cpu_p, allocation_size, hipMemcpyHostToDevice);		// cpu memory to gpu memory
	return result;
}


hipError_t gpuMemoryToCpuMemory()
{
	hipError_t result = hipMemcpy(cpu_p, gpu_p, allocation_size, hipMemcpyDeviceToHost);		// gpu memory to cpu memory
	return result;
}

void cpuFree()
{
	free(cpu_p);
}

hipError_t gpuFree()
{
	hipError_t result = hipFree(gpu_p);
	return result;
}

__global__ void gpuAdd(int* gpu_numbers)
{
	int threadId = threadIdx.x;

	gpu_numbers[threadId] *= 2;
}

void printCpuNumbers()
{
	int* cpu_int32 = (int*)cpu_p;

	for (size_t i = 0; i < number_count; i++) {
		printf("%d\t%d\n", i, cpu_int32[i]);
	}
}

void main()
{
	std::cout << cpuAlloc();
	cpuSetNumbers();

	std::cout << gpuAlloc();
	cpuMemoryToGpuMemory();

	// data process by gpu 

	gpuAdd << < 1, number_count >> > ((int*)gpu_p);
	hipError_t result = hipDeviceSynchronize();		// hipDeviceSynchronize -> waited all threads finish
	assert(result == hipSuccess);						// if it is result = 0, process successful  
		

	gpuMemoryToCpuMemory();

	printCpuNumbers();
	
	gpuFree();
	cpuFree();

	system("pause");
}