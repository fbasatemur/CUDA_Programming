#include "hip/hip_runtime.h"
#include ""

#include "stdio.h"
#include <stdlib.h>
#include <iostream>

const long allocation_size = 1 * 1024 * 1024 * 1024;	// 1GB allocation size

void* cpu_p;
void* gpu_p;

void cpuAlloc()											// cpu memory allocation
{
	cpu_p = malloc(allocation_size);
}

hipError_t gpuAlloc()
{
	hipError_t result = hipMalloc(&gpu_p, allocation_size);
	return result;
}

hipError_t gpuFree()
{
	hipError_t result = hipFree(gpu_p);
	return result;
}

void main()
{

	cpuAlloc();
	std::cout << gpuAlloc();							// result = 0 -> allocation success

	system("pause");

	try
	{
		std::cout << gpuFree();							// result = 0 -> flush success
		free(cpu_p);
	}
	catch (const std::exception & error)
	{
		std::cout << error.what();
	}

}