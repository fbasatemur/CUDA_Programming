#include "hip/hip_runtime.h"
#include ""

#include "stdio.h"
#include <stdlib.h>
#include <iostream>

const long allocation_size = 1 * 1024 * 1024 * 1024;

void* cpu_p;
void* gpu_p;

void cpuAlloc()
{
	cpu_p = malloc(allocation_size);
}

hipError_t gpuAlloc()
{
	hipError_t result = hipMalloc(&gpu_p, allocation_size);
	return result;
}

hipError_t gpuFree()
{
	hipError_t result = hipFree(gpu_p);
	return result;
}

void main()
{

	cpuAlloc();
	std::cout << gpuAlloc();		// return 0 -> true

	system("pause");

	try
	{
		gpuFree();
		free(cpu_p);
	}
	catch (const std::exception & error)
	{
		std::cout << error.what();
	}

}