#include "hip/hip_runtime.h"
#include<iostream>
#include<stdlib.h>

#include "hip/hip_runtime.h"
#include ""
#include <assert.h>

int number_count = 500 * 1024 * 1024;						// Allocation yapilacak int miktari
const int allocation_size = number_count * sizeof(int);		// number_count * 4 B

void* cpu_p;
void* gpu_p;



void cpuAlloc()												// allocation_size kadar RAM alani tahsis et
{
	cpu_p = malloc(allocation_size);
	assert(nullptr != cpu_p);
}

void gpuAlloc()												// allocation_size kadar GPU memory tahsis et
{
	hipError_t result = hipMalloc(&gpu_p, allocation_size);
	assert(result == hipSuccess);
}



void cpuMemoryToGpuMemory()									// cpu memory alanini (RAM), gpu memory alanina kopyala
{
	hipError_t result = hipMemcpy(gpu_p, cpu_p, allocation_size, hipMemcpyHostToDevice);		// cpu memory to gpu memory
	assert(result == hipSuccess);
}

void gpuMemoryToCpuMemory()									// gpu memory alanini, cpu memory alanina kopyala
{
	hipError_t result = hipMemcpy(cpu_p, gpu_p, allocation_size, hipMemcpyDeviceToHost);		// gpu memory to cpu memory
	assert(result == hipSuccess);
}


void cpuSetNumbers()										// cpu bellek alanina, number_count kadar sayi setle
{
	int* cpu_int32 = (int*)cpu_p;

	for (int i = 0; i < number_count; i++)
		cpu_int32[i] = i;

}

__global__ void gpuAdd(int* gpu_numbers)					// Paralel islemlenecek kisim, nvcc tarafindan burada compiler edilir
{
	//	int threadIndexOfTheThread = threadIdx.x;								// anlik thread index
	//	int blockIndexOfTheThread = blockIdx.x;									// anlik block index

	//	int threadCountInOneBlock = blockDim.x;									// bir bloktaki toplam thread sayisi
	//	int blockCountInThisKernel = gridDim.x;									// toplam block sayisi
	//	
	//	int id = blockIndexOfTheThread * threadCountInOneBlock + threadIndexOfTheThread;

	//	printf("%d \t %d \t  %d \t  %d \t  %d \t \n", id, threadIndexOfTheThread, blockIndexOfTheThread, blockCountInThisKernel, threadCountInOneBlock);

	
	// bir block sonlandiginda, thread index tekrar sifirlanacaktir.Bu ise veri karmasasina neden olabilir.
	int id = blockIdx.x * blockDim.x + threadIdx.x;			// Bunu engellemek icin index degeri duzenlenir

	gpu_numbers[id] *= 2;
}



void printCpuNumbers()										
{
	int* cpu_int32 = (int*)cpu_p;

	for (size_t i = number_count - 100; i < number_count; i++)		// son 100 degeri yazdir 
	{		
		printf("%d\t%d\n", i, cpu_int32[i]);
	}
}


void cpuFree()														// cpu memory serbest birak
{
	free(cpu_p);
}

void gpuFree()														// gpu memory serbest birak
{
	hipError_t result = hipFree(gpu_p);
	assert(result == hipSuccess);
}

void main()
{
	cpuAlloc();
	cpuSetNumbers();

	gpuAlloc();
	cpuMemoryToGpuMemory();

	// GPU bellegi uzerinden paralel veri islemleme yapiliyor..

	int blockDim = 64;									// Bir bloktaki toplam thread sayisi
	int gridDim = number_count / blockDim;				// toplam block sayisi = toplam thread sayisi / blockDim

	gpuAdd <<< gridDim, blockDim >> > ((int*)gpu_p);
	// GPU uzerinde tum islemler asenkron olarak yapilacaktir...
	hipError_t result = hipDeviceSynchronize();		// hipDeviceSynchronize ile tum islemlerin bitmesini bekleriz.. 
	assert(result == hipSuccess);						// if it is result = 0, process successful  


	gpuMemoryToCpuMemory();								// Cpu memory alanina, Gpu bellek alaninda islemlenen tum degerler aktarilir

	printCpuNumbers();

	gpuFree();
	cpuFree();

	getchar();
}